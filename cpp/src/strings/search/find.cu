#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/find.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/atomic>
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/find.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Threshold to decide on using string or warp parallel functions.
 *
 * If the average byte length of a string in a column exceeds this value then
 * a warp-parallel function is used.
 *
 * Note that this value is shared by find, rfind, and contains functions.
 */
constexpr size_type AVG_CHAR_BYTES_THRESHOLD = 64;

/**
 * @brief Find function handles a string per thread
 */
template <typename TargetIterator, bool forward = true>
struct finder_fn {
  column_device_view const d_strings;
  TargetIterator const d_targets;
  size_type const start;
  size_type const stop;

  __device__ size_type operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return -1; }
    auto const d_str = d_strings.element<string_view>(idx);
    if (d_str.empty() && (start > 0)) { return -1; }
    auto const d_target = d_targets[idx];

    auto const length = d_str.length();
    auto const begin  = (start > length) ? length : start;
    auto const end    = (stop < 0) || (stop > length) ? length : stop;
    return forward ? d_str.find(d_target, begin, end - begin)
                   : d_str.rfind(d_target, begin, end - begin);
  }
};

/**
 * @brief Special logic handles an empty target for find/rfind
 *
 * where length = number of characters in the input string
 * if forward = true:
 *   return start iff (start <= length), otherwise return -1
 * if forward = false:
 *   return stop iff (0 <= stop <= length), otherwise return length
 */
template <bool forward = true>
struct empty_target_fn {
  column_device_view const d_strings;
  size_type const start;
  size_type const stop;

  __device__ size_type operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return -1; }
    auto d_str = d_strings.element<string_view>(idx);

    // common case shortcut
    if (forward && start == 0) { return 0; }

    auto const length = d_str.length();
    if (start > length) { return -1; }
    if constexpr (forward) { return start; }

    return (stop < 0) || (stop > length) ? length : stop;
  }
};

/**
 * @brief String per warp function for find/rfind
 */
template <typename TargetIterator, bool forward = true>
CUDF_KERNEL void finder_warp_parallel_fn(column_device_view const d_strings,
                                         TargetIterator const d_targets,
                                         size_type const start,
                                         size_type const stop,
                                         size_type* d_results)
{
  size_type const idx = static_cast<size_type>(threadIdx.x + blockIdx.x * blockDim.x);

  if (idx >= (d_strings.size() * cudf::detail::warp_size)) { return; }

  auto const str_idx  = idx / cudf::detail::warp_size;
  auto const lane_idx = idx % cudf::detail::warp_size;

  if (d_strings.is_null(str_idx)) { return; }

  // initialize the output for the atomicMin/Max
  if (lane_idx == 0) { d_results[str_idx] = forward ? std::numeric_limits<size_type>::max() : -1; }
  __syncwarp();

  auto const d_str    = d_strings.element<string_view>(str_idx);
  auto const d_target = d_targets[str_idx];

  auto const [begin, left_over] = bytes_to_character_position(d_str, start);
  auto const start_char_pos     = start - left_over;  // keep track of character position

  auto const end = [d_str, start, stop, begin = begin] {
    if (stop < 0) { return d_str.size_bytes(); }
    if (stop <= start) { return begin; }
    // we count from `begin` instead of recounting from the beginning of the string
    return begin + std::get<0>(bytes_to_character_position(
                     string_view(d_str.data() + begin, d_str.size_bytes() - begin), stop - start));
  }();

  // each thread compares the target with the thread's individual starting byte
  size_type position = forward ? std::numeric_limits<size_type>::max() : -1;
  for (auto itr = begin + lane_idx; itr + d_target.size_bytes() <= end;
       itr += cudf::detail::warp_size) {
    if (d_target.compare(d_str.data() + itr, d_target.size_bytes()) == 0) {
      position = itr;
      if (forward) break;
    }
  }

  // find stores the minimum position while rfind stores the maximum position
  // note that this was slightly faster than using hipcub::WarpReduce
  cuda::atomic_ref<size_type, cuda::thread_scope_block> ref{*(d_results + str_idx)};
  forward ? ref.fetch_min(position, cuda::std::memory_order_relaxed)
          : ref.fetch_max(position, cuda::std::memory_order_relaxed);
  __syncwarp();

  if (lane_idx == 0) {
    // the final result needs to be fixed up convert max() to -1
    // and a byte position to a character position
    auto const result = d_results[str_idx];
    d_results[str_idx] =
      ((result < std::numeric_limits<size_type>::max()) && (result >= begin))
        ? start_char_pos + characters_in_string(d_str.data() + begin, result - begin)
        : -1;
  }
}

template <typename TargetIterator, bool forward = true>
void find_utility(strings_column_view const& input,
                  TargetIterator const& target_itr,
                  column& output,
                  size_type start,
                  size_type stop,
                  rmm::cuda_stream_view stream)
{
  auto d_strings = column_device_view::create(input.parent(), stream);
  auto d_results = output.mutable_view().data<size_type>();
  if ((input.chars_size(stream) / (input.size() - input.null_count())) > AVG_CHAR_BYTES_THRESHOLD) {
    // warp-per-string runs faster for longer strings (but not shorter ones)
    constexpr int block_size = 256;
    cudf::detail::grid_1d grid{input.size() * cudf::detail::warp_size, block_size};
    finder_warp_parallel_fn<TargetIterator, forward>
      <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
        *d_strings, target_itr, start, stop, d_results);
  } else {
    // string-per-thread function
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(input.size()),
                      d_results,
                      finder_fn<TargetIterator, forward>{*d_strings, target_itr, start, stop});
  }
}

template <bool forward = true>
std::unique_ptr<column> find_fn(strings_column_view const& input,
                                string_scalar const& target,
                                size_type start,
                                size_type stop,
                                rmm::cuda_stream_view stream,
                                rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(target.is_valid(stream), "Parameter target must be valid.");
  CUDF_EXPECTS(start >= 0, "Parameter start must be positive integer or zero.");
  if ((stop > 0) && (start > stop)) CUDF_FAIL("Parameter start must be less than stop.");

  // create output column
  auto results = make_numeric_column(data_type{type_to_id<size_type>()},
                                     input.size(),
                                     cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                     input.null_count(),
                                     stream,
                                     mr);
  // if input is empty or all-null then we are done
  if (input.size() == input.null_count()) { return results; }

  auto d_target = string_view(target.data(), target.size());

  // special logic for empty target results
  if (d_target.empty()) {
    auto d_strings = column_device_view::create(input.parent(), stream);
    auto d_results = results->mutable_view().data<size_type>();
    thrust::transform(rmm::exec_policy(stream),
                      thrust::counting_iterator<size_type>(0),
                      thrust::counting_iterator<size_type>(input.size()),
                      d_results,
                      empty_target_fn<forward>{*d_strings, start, stop});
    return results;
  }

  // find-utility function fills in the results column
  auto target_itr      = thrust::make_constant_iterator(d_target);
  using TargetIterator = decltype(target_itr);
  find_utility<TargetIterator, forward>(input, target_itr, *results, start, stop, stream);
  results->set_null_count(input.null_count());
  return results;
}
}  // namespace

std::unique_ptr<column> find(strings_column_view const& input,
                             string_scalar const& target,
                             size_type start,
                             size_type stop,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr)
{
  return find_fn<true>(input, target, start, stop, stream, mr);
}

std::unique_ptr<column> rfind(strings_column_view const& input,
                              string_scalar const& target,
                              size_type start,
                              size_type stop,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr)
{
  return find_fn<false>(input, target, start, stop, stream, mr);
}

template <bool forward = true>
std::unique_ptr<column> find(strings_column_view const& input,
                             strings_column_view const& target,
                             size_type start,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(start >= 0, "Parameter start must be positive integer or zero.");
  CUDF_EXPECTS(input.size() == target.size(), "input and target columns must be the same size");

  // create output column
  auto results = make_numeric_column(
    data_type{type_to_id<size_type>()}, input.size(), rmm::device_buffer{}, 0, stream, mr);
  // if input is empty or all-null then we are done
  if (input.size() == input.null_count()) { return results; }

  // call find utility with target iterator
  auto d_targets  = column_device_view::create(target.parent(), stream);
  auto target_itr = cudf::detail::make_null_replacement_iterator<string_view>(
    *d_targets, string_view{}, target.has_nulls());
  find_utility<decltype(target_itr), forward>(input, target_itr, *results, start, -1, stream);

  // AND the bitmasks from input and target
  auto [null_mask, null_count] =
    cudf::detail::bitmask_and(table_view({input.parent(), target.parent()}), stream, mr);
  results->set_null_mask(std::move(null_mask), null_count);
  return results;
}

}  // namespace detail

// external APIs

std::unique_ptr<column> find(strings_column_view const& strings,
                             string_scalar const& target,
                             size_type start,
                             size_type stop,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::find(strings, target, start, stop, stream, mr);
}

std::unique_ptr<column> rfind(strings_column_view const& strings,
                              string_scalar const& target,
                              size_type start,
                              size_type stop,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::rfind(strings, target, start, stop, stream, mr);
}

std::unique_ptr<column> find(strings_column_view const& input,
                             strings_column_view const& target,
                             size_type start,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::find<true>(input, target, start, stream, mr);
}

namespace detail {
namespace {

/**
 * @brief Check if `d_target` appears in a row in `d_strings`.
 *
 * This executes as a warp per string/row and performs well for longer strings.
 * @see AVG_CHAR_BYTES_THRESHOLD
 *
 * @param d_strings Column of input strings
 * @param d_target String to search for in each row of `d_strings`
 * @param d_results Indicates which rows contain `d_target`
 */
CUDF_KERNEL void contains_warp_parallel_fn(column_device_view const d_strings,
                                           string_view const d_target,
                                           bool* d_results)
{
  size_type const idx = static_cast<size_type>(threadIdx.x + blockIdx.x * blockDim.x);
  using warp_reduce   = hipcub::WarpReduce<bool>;
  __shared__ typename warp_reduce::TempStorage temp_storage;

  if (idx >= (d_strings.size() * cudf::detail::warp_size)) { return; }

  auto const str_idx  = idx / cudf::detail::warp_size;
  auto const lane_idx = idx % cudf::detail::warp_size;
  if (d_strings.is_null(str_idx)) { return; }
  // get the string for this warp
  auto const d_str = d_strings.element<string_view>(str_idx);
  // each warp processes 4 starting bytes
  auto constexpr bytes_per_warp = 4;
  auto found                    = false;
  for (auto i = lane_idx * bytes_per_warp;
       !found && ((i + d_target.size_bytes()) <= d_str.size_bytes());
       i += cudf::detail::warp_size * bytes_per_warp) {
    // check the target matches this part of the d_str data
    // this is definitely faster for very long strings > 128B
    for (auto j = 0; j < bytes_per_warp; j++) {
      if (((i + j + d_target.size_bytes()) <= d_str.size_bytes()) &&
          d_target.compare(d_str.data() + i + j, d_target.size_bytes()) == 0) {
        found = true;
      }
    }
  }

  auto const result = warp_reduce(temp_storage).Reduce(found, hipcub::Max());
  if (lane_idx == 0) { d_results[str_idx] = result; }
}

std::unique_ptr<column> contains_warp_parallel(strings_column_view const& input,
                                               string_scalar const& target,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(target.is_valid(stream), "Parameter target must be valid.");
  auto d_target = string_view(target.data(), target.size());

  // create output column
  auto results = make_numeric_column(data_type{type_id::BOOL8},
                                     input.size(),
                                     cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                     input.null_count(),
                                     stream,
                                     mr);

  // fill the output with `false` unless the `d_target` is empty
  auto results_view = results->mutable_view();
  if (d_target.empty()) {
    thrust::fill(
      rmm::exec_policy_nosync(stream), results_view.begin<bool>(), results_view.end<bool>(), true);
  } else {
    // launch warp per string
    auto const d_strings     = column_device_view::create(input.parent(), stream);
    constexpr int block_size = 256;
    cudf::detail::grid_1d grid{input.size() * cudf::detail::warp_size, block_size};
    contains_warp_parallel_fn<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *d_strings, d_target, results_view.data<bool>());
  }
  results->set_null_count(input.null_count());
  return results;
}

/**
 * Each string uses a warp(32 threads) to handle all the targets.
 * Each thread uses num_targets bools shared memory to store temp result for each lane.
 */
CUDF_KERNEL void multi_contains_warp_parallel_multi_scalars_fn(
  column_device_view const d_strings,
  column_device_view const d_targets,
  cudf::device_span<char> const d_target_first_bytes,
  column_device_view const d_target_indexes_for_first_bytes,
  cudf::device_span<bool*> d_results)
{
  auto const num_targets = d_targets.size();
  auto const num_rows    = d_strings.size();

  auto const idx = static_cast<size_type>(threadIdx.x + blockIdx.x * blockDim.x);
  if (idx >= (num_rows * cudf::detail::warp_size)) { return; }

  auto const lane_idx = idx % cudf::detail::warp_size;
  auto const str_idx  = idx / cudf::detail::warp_size;
  if (d_strings.is_null(str_idx)) { return; }  // bitmask will set result to null.
  // get the string for this warp
  auto const d_str = d_strings.element<string_view>(str_idx);

  /**
   * size of shared_bools = Min(targets_size * block_size, target_group * block_size)
   * each thread uses targets_size bools
   */
  extern __shared__ bool shared_bools[];

  // initialize temp result:
  // set true if target is empty, set false otherwise
  for (int target_idx = 0; target_idx < num_targets; target_idx++) {
    auto const d_target = d_targets.element<string_view>(target_idx);
    shared_bools[threadIdx.x * num_targets + target_idx] = d_target.size_bytes() == 0;
  }

  for (size_type str_byte_idx = lane_idx; str_byte_idx < d_str.size_bytes();
       str_byte_idx += cudf::detail::warp_size) {
    // 1. check the first chars using binary search on first char set
    char c = *(d_str.data() + str_byte_idx);
    auto first_byte_ptr =
      thrust::lower_bound(thrust::seq, d_target_first_bytes.begin(), d_target_first_bytes.end(), c);
    if (not(first_byte_ptr != d_target_first_bytes.end() && *first_byte_ptr == c)) {
      // first char is not matched for all targets, already set result as found
      continue;
    }

    // 2. check the 2nd chars
    int first_char_index_in_list = first_byte_ptr - d_target_first_bytes.begin();
    // get possible targets
    auto const possible_targets_list =
      cudf::list_device_view{d_target_indexes_for_first_bytes, first_char_index_in_list};
    for (auto list_idx = 0; list_idx < possible_targets_list.size();
         ++list_idx) {  // iterate possible targets
      auto target_idx     = possible_targets_list.element<size_type>(list_idx);
      int temp_result_idx = threadIdx.x * num_targets + target_idx;
      if (!shared_bools[temp_result_idx]) {  // not found before
        auto const d_target = d_targets.element<string_view>(target_idx);
        if (d_str.size_bytes() - str_byte_idx >= d_target.size_bytes()) {
          // first char already checked, only need to check the [2nd, end) chars if has.
          bool found = true;
          for (auto i = 1; i < d_target.size_bytes(); i++) {
            if (*(d_str.data() + str_byte_idx + i) != *(d_target.data() + i)) {
              found = false;
              break;
            }
          }
          if (found) { shared_bools[temp_result_idx] = true; }
        }
      }
    }
  }

  // wait all lanes are done in a warp
  __syncwarp();

  if (lane_idx == 0) {
    for (int target_idx = 0; target_idx < num_targets; target_idx++) {
      bool found = false;
      for (int lane_idx = 0; lane_idx < cudf::detail::warp_size; lane_idx++) {
        bool temp_idx = (str_idx * cudf::detail::warp_size + lane_idx) * num_targets + target_idx;
        if (shared_bools[temp_idx]) {
          found = true;
          break;
        }
      }
      d_results[target_idx][str_idx] = found;
    }
  }
}

CUDF_KERNEL void multi_contains_using_indexes_fn(
  column_device_view const d_strings,
  column_device_view const d_targets,
  cudf::device_span<char> const d_target_first_bytes,
  column_device_view const d_target_indexes_for_first_bytes,
  cudf::device_span<bool*> d_results)
{
  auto const str_idx     = static_cast<size_type>(cudf::detail::grid_1d::global_thread_id());
  auto const num_targets = d_targets.size();
  auto const num_rows    = d_strings.size();
  if (str_idx >= num_rows) { return; }
  if (d_strings.is_null(str_idx)) { return; }  // bitmask will set result to null.
  auto const d_str = d_strings.element<string_view>(str_idx);

  // check empty target, the result of searching empty target is true.
  for (auto target_idx = 0; target_idx < num_targets; ++target_idx) {
    auto const d_target            = d_targets.element<string_view>(target_idx);
    d_results[target_idx][str_idx] = d_target.size_bytes() == 0;
  }

  for (auto str_byte_idx = 0; str_byte_idx < d_str.size_bytes();
       ++str_byte_idx) {  // iterate the start index in the string

    // 1. check the first chars using binary search on first char set
    char c = *(d_str.data() + str_byte_idx);
    auto first_byte_ptr =
      thrust::lower_bound(thrust::seq, d_target_first_bytes.begin(), d_target_first_bytes.end(), c);

    if (not(first_byte_ptr != d_target_first_bytes.end() && *first_byte_ptr == c)) {
      // For non-empty targets: no need to search for `str_byte_idx` position, because first char is
      // unmatched. For empty targets: already set result as found.
      continue;
    }

    int first_char_index_in_list = first_byte_ptr - d_target_first_bytes.begin();
    // get possible targets
    auto const possible_targets_list =
      cudf::list_device_view{d_target_indexes_for_first_bytes, first_char_index_in_list};

    for (auto list_idx = 0; list_idx < possible_targets_list.size();
         ++list_idx) {  // iterate possible targets
      auto target_idx = possible_targets_list.element<size_type>(list_idx);
      if (!d_results[target_idx][str_idx]) {  // not found before
        auto const d_target = d_targets.element<string_view>(target_idx);
        if (d_str.size_bytes() - str_byte_idx >= d_target.size_bytes()) {
          // first char already checked, only need to check the [2nd, end) chars if has.
          bool found = true;
          for (auto i = 1; i < d_target.size_bytes(); i++) {
            if (*(d_str.data() + str_byte_idx + i) != *(d_target.data() + i)) {
              found = false;
              break;
            }
          }
          if (found) { d_results[target_idx][str_idx] = true; }
        }
      }
    }
  }
}

/**
 * Execute multi contains for short strings
 * First index the first char for all targets.
 * Index the first char:
 *   collect first char for all targets and do uniq and sort,
 *   then index the targets for the first char.
 *   e.g.:
 *     targets: xa xb ac ad af
 *     first char set is: (a, x)
 *     index result is:
 *       {
 *         a: [2, 3, 4],   // indexes for: ac ad af
 *         x: [0, 1]       // indexes for: xa xb
 *       }
 * when do searching:
 *   find (binary search) from `first char set` for a char in string:
 *     if char in string is not in ['a', 'x'], fast skip
 *     if char in string is 'x', then only need to try ["xa", "xb"] targets.
 *     if char in string is 'a', then only need to try ["ac", "ad", "af"] targets.
 *
 */
std::vector<std::unique_ptr<column>> multi_contains(bool warp_parallel,
                                                    strings_column_view const& input,
                                                    strings_column_view const& targets,
                                                    rmm::cuda_stream_view stream,
                                                    rmm::mr::device_memory_resource* mr)
{
  auto const num_targets = static_cast<size_type>(targets.size());
  CUDF_EXPECTS(not targets.is_empty(), "Must specify at least one target string.");

  // 1. copy targets from device to host
  auto const h_targets_child = cudf::detail::make_std_vector_sync<char>(
    cudf::device_span<char const>(targets.chars_begin(stream), targets.chars_size(stream)), stream);
  auto const targets_offsets   = targets.offsets();
  auto const h_targets_offsets = cudf::detail::make_std_vector_sync(
    cudf::device_span<int const>{targets_offsets.data<int>(),
                                 static_cast<size_t>(targets_offsets.size())},
    stream);

  // 2. index the first characters in targets
  // 2.1 collect first characters in targets
  thrust::host_vector<char> h_first_bytes = {};
  for (auto i = 0; i < targets.size(); i++) {
    auto target_begin_offset = h_targets_offsets[i];
    auto target_end_offset   = h_targets_offsets[i + 1];
    if (target_end_offset - target_begin_offset > 0) {
      char first_char = h_targets_child[target_begin_offset];
      auto no_exist =
        thrust::find(h_first_bytes.begin(), h_first_bytes.end(), first_char) == h_first_bytes.end();
      if (no_exist) { h_first_bytes.push_back(first_char); }
    }
  }

  // 2.2 sort the first characters
  thrust::sort(h_first_bytes.begin(), h_first_bytes.end());

  // 2.3 generate indexes: map from `first char in target` to `target indexes`
  thrust::host_vector<size_type> h_offsets  = {0};
  thrust::host_vector<size_type> h_elements = {};
  for (size_t i = 0; i < h_first_bytes.size(); i++) {
    auto expected_first_byte = h_first_bytes[i];
    for (auto target_idx = 0; target_idx < targets.size(); target_idx++) {
      auto target_begin_offset = h_targets_offsets[target_idx];
      auto target_end_offset   = h_targets_offsets[target_idx + 1];
      if (target_end_offset - target_begin_offset > 0) {
        char curr_first_byte = h_targets_child[target_begin_offset];
        if (expected_first_byte == curr_first_byte) { h_elements.push_back(target_idx); }
      }
    }
    h_offsets.push_back(h_elements.size());
  }

  // 2.4 copy first char set and first char indexes to device
  auto d_first_bytes  = cudf::detail::make_device_uvector_async(h_first_bytes, stream, mr);
  auto d_offsets      = cudf::detail::make_device_uvector_async(h_offsets, stream, mr);
  auto d_elements     = cudf::detail::make_device_uvector_async(h_elements, stream, mr);
  auto offsets_column = std::make_unique<cudf::column>(cudf::data_type{cudf::type_id::INT32},
                                                       h_offsets.size(),
                                                       d_offsets.release(),
                                                       rmm::device_buffer{},  // null mask
                                                       0                      // null size
  );
  auto element_column = std::make_unique<cudf::column>(cudf::data_type{cudf::type_id::INT32},
                                                       h_elements.size(),
                                                       d_elements.release(),
                                                       rmm::device_buffer{},  // null mask
                                                       0                      // null size
  );
  auto list_column    = cudf::make_lists_column(h_first_bytes.size(),
                                             std::move(offsets_column),
                                             std::move(element_column),
                                             0,                     // null count
                                             rmm::device_buffer{},  // null mask
                                             stream,
                                             mr);
  auto d_list_column  = column_device_view::create(list_column->view(), stream);

  // 3. Create output columns.
  auto const results_iter =
    thrust::make_transform_iterator(thrust::counting_iterator<cudf::size_type>(0), [&](int i) {
      return make_numeric_column(data_type{type_id::BOOL8},
                                 input.size(),
                                 cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                 input.null_count(),
                                 stream,
                                 mr);
    });
  auto results_list =
    std::vector<std::unique_ptr<column>>(results_iter, results_iter + targets.size());
  auto device_results_list = [&] {
    auto host_results_pointer_iter =
      thrust::make_transform_iterator(results_list.begin(), [](auto const& results_column) {
        return results_column->mutable_view().template data<bool>();
      });
    auto host_results_pointers = std::vector<bool*>(
      host_results_pointer_iter, host_results_pointer_iter + results_list.size());
    return cudf::detail::make_device_uvector_async(host_results_pointers, stream, mr);
  }();

  auto const d_strings = column_device_view::create(input.parent(), stream);
  auto const d_targets = column_device_view::create(targets.parent(), stream);

  constexpr int block_size = 256;
  cudf::detail::grid_1d grid{input.size(), block_size};

  if (warp_parallel) {
    int shared_mem_size = block_size * targets.size();
    multi_contains_warp_parallel_multi_scalars_fn<<<grid.num_blocks,
                                                    grid.num_threads_per_block,
                                                    shared_mem_size,
                                                    stream.value()>>>(
      *d_strings, *d_targets, d_first_bytes, *d_list_column, device_results_list);
  } else {
    multi_contains_using_indexes_fn<<<grid.num_blocks,
                                      grid.num_threads_per_block,
                                      0,
                                      stream.value()>>>(
      *d_strings, *d_targets, d_first_bytes, *d_list_column, device_results_list);
  }

  return results_list;
}

/**
 * @brief Utility to return a bool column indicating the presence of
 * a given target string in a strings column.
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam BoolFunction Return bool value given two strings.
 *
 * @param strings Column of strings to check for target.
 * @param target UTF-8 encoded string to check in strings column.
 * @param pfn Returns bool value if target is found in the given string.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @return New BOOL column.
 */
template <typename BoolFunction>
std::unique_ptr<column> contains_fn(strings_column_view const& strings,
                                    string_scalar const& target,
                                    BoolFunction pfn,
                                    rmm::cuda_stream_view stream,
                                    rmm::device_async_resource_ref mr)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(type_id::BOOL8);

  CUDF_EXPECTS(target.is_valid(stream), "Parameter target must be valid.");
  if (target.size() == 0)  // empty target string returns true
  {
    auto const true_scalar = make_fixed_width_scalar<bool>(true, stream);
    auto results           = make_column_from_scalar(*true_scalar, strings.size(), stream, mr);
    results->set_null_mask(cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                           strings.null_count());
    return results;
  }

  auto d_target       = string_view(target.data(), target.size());
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create output column
  auto results      = make_numeric_column(data_type{type_id::BOOL8},
                                     strings_count,
                                     cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<bool>();
  // set the bool values by evaluating the passed function
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_results,
                    [d_strings, pfn, d_target] __device__(size_type idx) {
                      return !d_strings.is_null(idx) &&
                             bool{pfn(d_strings.element<string_view>(idx), d_target)};
                    });
  results->set_null_count(strings.null_count());
  return results;
}

/**
 * @brief Utility to return a bool column indicating the presence of
 * a string targets[i] in strings[i].
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam BoolFunction Return bool value given two strings.
 *
 * @param strings Column of strings to check for `targets[i]`.
 * @param targets Column of strings to be checked in `strings[i]``.
 * @param pfn Returns bool value if target is found in the given string.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @return New BOOL column.
 */
template <typename BoolFunction>
std::unique_ptr<column> contains_fn(strings_column_view const& strings,
                                    strings_column_view const& targets,
                                    BoolFunction pfn,
                                    rmm::cuda_stream_view stream,
                                    rmm::device_async_resource_ref mr)
{
  if (strings.is_empty()) return make_empty_column(type_id::BOOL8);

  CUDF_EXPECTS(targets.size() == strings.size(),
               "strings and targets column must be the same size");

  auto targets_column = column_device_view::create(targets.parent(), stream);
  auto d_targets      = *targets_column;
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create output column
  auto results      = make_numeric_column(data_type{type_id::BOOL8},
                                     strings.size(),
                                     cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<bool>();
  // set the bool values by evaluating the passed function
  thrust::transform(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings.size()),
    d_results,
    [d_strings, pfn, d_targets] __device__(size_type idx) {
      // empty target string returns true
      if (d_targets.is_valid(idx) && d_targets.element<string_view>(idx).length() == 0) {
        return true;
      } else if (!d_strings.is_null(idx) && !d_targets.is_null(idx)) {
        return bool{pfn(d_strings.element<string_view>(idx), d_targets.element<string_view>(idx))};
      } else {
        return false;
      }
    });
  results->set_null_count(strings.null_count());
  return results;
}

std::unique_ptr<column> contains_small_strings_impl(strings_column_view const& input,
                                                    string_scalar const& target,
                                                    rmm::cuda_stream_view stream,
                                                    rmm::device_async_resource_ref mr)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return d_string.find(d_target) != string_view::npos;
  };
  return contains_fn(input, target, pfn, stream, mr);
}
}  // namespace

std::unique_ptr<column> contains(strings_column_view const& input,
                                 string_scalar const& target,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
{
  // use warp parallel when the average string width is greater than the threshold
  if ((input.null_count() < input.size()) &&
      ((input.chars_size(stream) / input.size()) > AVG_CHAR_BYTES_THRESHOLD)) {
    return contains_warp_parallel(input, target, stream, mr);
  }

  // benchmark measurements showed this to be faster for smaller strings
  return contains_small_strings_impl(input, target, stream, mr);
}

std::unique_ptr<table> multi_contains(strings_column_view const& input,
                                      strings_column_view const& targets,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(not targets.has_nulls(), "Target strings cannot be null");
  auto result_columns = [&] {
    if ((input.null_count() < input.size()) &&
        ((input.chars_size(stream) / input.size()) > AVG_CHAR_BYTES_THRESHOLD)) {
      // Large strings.
      // use warp parallel when the average string width is greater than the threshold
      return multi_contains(/**warp parallel**/ true, input, targets, stream, mr);
    } else {
      // Small strings. Searching for multiple targets in one thread seems to work fastest.
      return multi_contains(/**warp parallel**/ false, input, targets, stream, mr);
    }
  }();
  return std::make_unique<table>(std::move(result_columns));
}

std::unique_ptr<column> contains(strings_column_view const& strings,
                                 strings_column_view const& targets,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return d_string.find(d_target) != string_view::npos;
  };
  return contains_fn(strings, targets, pfn, stream, mr);
}

std::unique_ptr<column> starts_with(strings_column_view const& strings,
                                    string_scalar const& target,
                                    rmm::cuda_stream_view stream,
                                    rmm::device_async_resource_ref mr)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return (d_target.size_bytes() <= d_string.size_bytes()) &&
           (d_target.compare(d_string.data(), d_target.size_bytes()) == 0);
  };
  return contains_fn(strings, target, pfn, stream, mr);
}

std::unique_ptr<column> starts_with(strings_column_view const& strings,
                                    strings_column_view const& targets,
                                    rmm::cuda_stream_view stream,
                                    rmm::device_async_resource_ref mr)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return (d_target.size_bytes() <= d_string.size_bytes()) &&
           (d_target.compare(d_string.data(), d_target.size_bytes()) == 0);
  };
  return contains_fn(strings, targets, pfn, stream, mr);
}

std::unique_ptr<column> ends_with(strings_column_view const& strings,
                                  string_scalar const& target,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    auto const str_size = d_string.size_bytes();
    auto const tgt_size = d_target.size_bytes();
    return (tgt_size <= str_size) &&
           (d_target.compare(d_string.data() + str_size - tgt_size, tgt_size) == 0);
  };

  return contains_fn(strings, target, pfn, stream, mr);
}

std::unique_ptr<column> ends_with(strings_column_view const& strings,
                                  strings_column_view const& targets,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    auto const str_size = d_string.size_bytes();
    auto const tgt_size = d_target.size_bytes();
    return (tgt_size <= str_size) &&
           (d_target.compare(d_string.data() + str_size - tgt_size, tgt_size) == 0);
  };

  return contains_fn(strings, targets, pfn, stream, mr);
}

}  // namespace detail

// external APIs

std::unique_ptr<column> contains(strings_column_view const& strings,
                                 string_scalar const& target,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(strings, target, stream, mr);
}

std::unique_ptr<table> multi_contains(strings_column_view const& strings,
                                      strings_column_view const& targets,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::multi_contains(strings, targets, stream, mr);
}

std::unique_ptr<column> contains(strings_column_view const& strings,
                                 strings_column_view const& targets,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(strings, targets, stream, mr);
}

std::unique_ptr<column> starts_with(strings_column_view const& strings,
                                    string_scalar const& target,
                                    rmm::cuda_stream_view stream,
                                    rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::starts_with(strings, target, stream, mr);
}

std::unique_ptr<column> starts_with(strings_column_view const& strings,
                                    strings_column_view const& targets,
                                    rmm::cuda_stream_view stream,
                                    rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::starts_with(strings, targets, stream, mr);
}

std::unique_ptr<column> ends_with(strings_column_view const& strings,
                                  string_scalar const& target,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::ends_with(strings, target, stream, mr);
}

std::unique_ptr<column> ends_with(strings_column_view const& strings,
                                  strings_column_view const& targets,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::ends_with(strings, targets, stream, mr);
}

}  // namespace strings
}  // namespace cudf
